
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <cassert>
#include <chrono>
#include <random>
#include <string>
#include <ctime>
#include <algorithm>
#include <fcntl.h>
#include <unistd.h>

//constants
#define TRAINING_SET_SIZE 60000
#define TEST_SET_SIZE 10000
#define COLS 28
#define ROWS 28
#define IMG_MAGIC_NUM 0x00000803
#define LABEL_MAGIC_NUM 0x00000801
#define NUM_LABELS 60000
#define NUM_NEURONS 1024
#define EPOCH_SIZE 100
#define BATCH_SIZE 100

//All the integers in the files are stored in the MSB first (high endian) format
void toLittleEndian(int &num){
    num = (0xFF&(num >> 24))      |
          (0xFF00&(num >> 8))     |
          (0xFF0000&(num << 8))   |
          (0xFF000000&(num << 24));
}

void read_images(const std::string &file_name, float*** (&imgs)){
    int fd;
    fd = open(file_name.c_str(), O_RDONLY);
    assert(fd >= 0);

    int rv, magic_num, num_imgs, num_cols, num_rows;

    rv = read(fd, &magic_num, 4);
    assert(rv == 4);
    //change endianess
    toLittleEndian(magic_num);
    assert(magic_num == 0x803);

    rv = read(fd, &num_imgs, 4);
    assert(rv == 4);
    //change endianess
    toLittleEndian(num_imgs);

    rv = read(fd, &num_rows, 4);
    assert(rv == 4);
    //change endianness
    toLittleEndian(num_rows);

    rv = read(fd, &num_cols, 4);
    assert(rv == 4);
    //change endianness
    toLittleEndian(num_cols);

    imgs = new float**[num_imgs]();
    for(int i = 0; i < num_imgs; i++){
        imgs[i] = new float*[num_rows]();

        //read whole image at once to minimize IO since that takes time
        unsigned char tmp_img[num_rows][num_cols];
        rv = read(fd, tmp_img, num_rows*num_cols);
        assert(rv == num_rows*num_cols);

        for(int r = 0; r < num_rows; r++){
            imgs[i][r] = new float[num_cols]();
            for(int c = 0; c < num_cols; c++){
                imgs[i][r][c] = double(tmp_img[r][c])/127.5 - 1;
            }
        }

    }
    rv = close(fd);
    assert(rv == 0);
}

void read_labels(const std::string &file_name, unsigned char* (&labels)){
    int fd;
    fd = open(file_name.c_str(), O_RDONLY);
    assert(fd >= 0);

    int magic_num, num_labels;

    int rv = read(fd, &magic_num, 4);
    assert(rv == 4);
    //change endianess
    toLittleEndian(magic_num);

    rv = read(fd, &num_labels, 4);
    assert(rv == 4);
    //change endianess
    toLittleEndian(num_labels);

    labels = new unsigned char[num_labels]();
    rv = read(fd, labels, num_labels);
    for(int i = 0; i < num_labels; i++){
        //all labels are 0-9
        //assert(labels[i] >= 0 && labels[i] <= 9);
    }
    rv = close(fd);
    assert(rv == 0);

}

void generateWeights(float*** (&ilw), float*** (&ild), float** (&fclw), float** (&fcld)){

    //unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    unsigned seed = 8493;
    std::default_random_engine generator(seed);
    std::normal_distribution<float> distribution;

    ilw = new float**[NUM_NEURONS]();
    ild = new float**[NUM_NEURONS]();
    fclw = new float*[(int) NUM_NEURONS/EPOCH_SIZE]();
    fcld = new float*[(int) NUM_NEURONS/EPOCH_SIZE]();

    for(int n = 0; n < NUM_NEURONS; n++){
        ilw[n] = new float*[ROWS]();
        ild[n] = new float*[ROWS]();
        for(int r = 0; r < ROWS; r++){
            ilw[n][r] = new float[COLS]();
            ild[n][r] = new float[COLS]();
            for(int c = 0; c < COLS; c++){
                //normal_distribution represents unbownded distribution, divide by sqrt(N)
                ilw[n][r][c] = distribution(generator) / sqrt(NUM_NEURONS);
                //std::cout << distribution(generator) / sqrt(NUM_NEURONS) << std::endl;
                //initially weights are 0
                ild[n][r][c] = 0;
            }
        }
    }

    for(int i = 0; i < (int) NUM_NEURONS/EPOCH_SIZE; i++){
        fclw[i] = new float[NUM_NEURONS]();
        fcld[i] = new float[NUM_NEURONS]();
        for (int n = 0; n < NUM_NEURONS; n++){
            fclw[i][n] = distribution(generator) / sqrt((int) NUM_NEURONS/EPOCH_SIZE);
            fcld[i][n] = 0;
        }
    }
}

//based on softmax from prof. Chiu's examples
float* softmax(float *in){
    // Use identity softmax(x) == softmax(x - C)
    const auto C = *std::max_element(in, in+((int) NUM_NEURONS/EPOCH_SIZE));
    //std::cout << "Max element: " << C << std::endl;
    //same length as in
    float* out = new float[(int) NUM_NEURONS/EPOCH_SIZE];
    float sum = 0;
    for(size_t i = 0; i < (int) NUM_NEURONS/EPOCH_SIZE; i++){
        //std::cout << in[i] - C << std::endl;
        out[i] = std::exp(in[i] - C);
        //assert(out[i] != 0);
        sum += out[i];
    }
    /*
    for(size_t i = 0; i < (int) NUM_NEURONS/EPOCH_SIZE; i++){
        out[i] = out[i]/sum;
    }
    */
    // for(size_t i = 0; i < (int) NUM_NEURONS/EPOCH_SIZE; i++){
    //     assert(out[i] != 0);
    //     std::cout << "GOOD" << std::endl;
    // }
    std::transform(out, out + ((int) NUM_NEURONS/EPOCH_SIZE), out, [sum](float e) {return e/sum;});

    // for(size_t i = 0; i < (int) NUM_NEURONS/EPOCH_SIZE; i++){
    //     assert(out[i] != 0);
    //     if(out[i] != 0){
    //         std::cout << out[i] << std::endl;
    //     }
    // }

    return out;

}

float* softmax_ds(float* out, float* us){
    float* sm_ds = new float[(int) NUM_NEURONS/EPOCH_SIZE]();
    for(size_t i = 0; i < (int) NUM_NEURONS/EPOCH_SIZE; i++){
        for(size_t j = 0; j < (int) NUM_NEURONS/EPOCH_SIZE; j++){
            if( i == j) {
                sm_ds[i] += (out[j]*(1 - out[i])) * us[j];
            } else {
                sm_ds[i] += (-out[i]*out[j])*us[j];
            }
        }
    }
    return sm_ds;
}

__global__ void update_dense_weights(float *w1, float *ds1){
    //current thread and node num
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    w1[tid] -= (BATCH_SIZE/1000)*ds1[tid];
    ds1[tid] = 0;

}

int main(int argc, char** argv){
    if(argc != 3){
        std::cerr << "Wrong number of inputs. Usage: ./parallelNN <images> <labels>" << std::endl;
        exit(1);
    }

    //read training
    static float ***training_images;
    static unsigned char *training_labels;
    read_images(std::string(argv[1]), training_images);
    read_labels(std::string(argv[2]), training_labels);

    float ***input_layer_w, ***input_layer_ds;
    float **fully_connected_layer_w, **fully_connected_layer_ds;

    //std::cout << input_layer_w[0][0][0] << std::endl;

    generateWeights(input_layer_w, input_layer_ds, fully_connected_layer_w, fully_connected_layer_ds);

    //First fully connected layer
    float *first_layer = new float[NUM_NEURONS]();
    float *first_layer_ds = new float[NUM_NEURONS]();

    //Second fully connected layer
    float *second_layer = new float[(int)NUM_NEURONS/EPOCH_SIZE]();
    float *second_layer_ds = new float[NUM_NEURONS]();

    //Softmax layer
    float *soft_max_layer = new float[(int)NUM_NEURONS/EPOCH_SIZE]();
    float *soft_max_layer_ds = new float[(int)NUM_NEURONS/EPOCH_SIZE]();

    //Cross-entropy layer
    float* cross_ent_layer = new float[(int)NUM_NEURONS/EPOCH_SIZE]();

    //CUDA
    float *dense_layer_w1, *dense_layer_ds1, *dense_layer_w2, *dense_layer_ds2;
    //place them in contiguoys memory
    float *hidden_layer_w1 = new float[NUM_NEURONS*ROWS*COLS]();
    float *hidden_layer_ds1 = new float[NUM_NEURONS*ROWS*COLS]();
    float *hidden_layer_w2 = new float[NUM_NEURONS* ((int)NUM_NEURONS/EPOCH_SIZE)]();
    float *hidden_layer_ds2 = new float[NUM_NEURONS* ((int)NUM_NEURONS/EPOCH_SIZE)]();

    hipMalloc(&dense_layer_w1, NUM_NEURONS*ROWS*COLS*(sizeof(float)));
    hipMalloc(&dense_layer_ds1, NUM_NEURONS*ROWS*COLS*(sizeof(float)));
    hipMalloc(&dense_layer_w2, NUM_NEURONS* ((int)NUM_NEURONS/EPOCH_SIZE) *(sizeof(float)));
    hipMalloc(&dense_layer_ds2, NUM_NEURONS* ((int)NUM_NEURONS/EPOCH_SIZE) *(sizeof(float)));

    //to generate random number for dropout
    std::srand(std::time(0));

    for(int e = 0; e < EPOCH_SIZE; e++){
        //rounds
        for(int j = 0; j < EPOCH_SIZE; j++){

            //FORWARD

            //initialize values
            int correct = 0, total = 0;

            //loop through images in batch
            for(int i = 0; i < BATCH_SIZE; i++){
                for(int k = 0; k < (int) NUM_NEURONS/EPOCH_SIZE; k++){
                    cross_ent_layer[k] = 0;
                }
                //current label and img displaced by i (the images already processed)
                int current_label = (int) training_labels[EPOCH_SIZE*j + i];
                float** current_image = training_images[EPOCH_SIZE*j + i];

                for(int n = 0; n < NUM_NEURONS; n++){
                    float temp_result = 0;
                    //dropout rate of 0.4%
                    if(std::rand() % 1000 < 4){
                        first_layer[n] = 0;
                    } else{
                        for(int r = 0; r < ROWS; r++){
                            for(int c = 0; c < COLS; c++){
                                //calculate results of the first layer
                                temp_result += input_layer_w[n][r][c] * current_image[r][c];
                            }
                        }
                        //ReLU
                        if(temp_result < 0){
                            first_layer[n] = 0;
                        } else{
                            first_layer[n] = temp_result;
                        }
                    }
                }
                //std::cout << "1" << std::endl;
                //std::cout << input_layer_w[0][0][0] << std::endl;

                for(int k = 0; k < (int) NUM_NEURONS/EPOCH_SIZE; k++){
                    for(int n = 0; n < NUM_NEURONS; n++){
                        //second_layer weights are too large/small
                        second_layer[k] += fully_connected_layer_w[k][n] * first_layer[n];
                    }
                }

                soft_max_layer = softmax(second_layer);
                if(std::distance(soft_max_layer, std::max_element(soft_max_layer, soft_max_layer+(int) NUM_NEURONS/EPOCH_SIZE)) == current_label){
                    correct++;
                }
                total++;

                cross_ent_layer[current_label] = -1 / soft_max_layer[current_label];

                //BACK-PROPAGATION

                soft_max_layer_ds = softmax_ds(soft_max_layer, cross_ent_layer);

                for(int k = 0; k < (int) NUM_NEURONS/EPOCH_SIZE; k++){
                    for(int n = 0; n < NUM_NEURONS; n++){
                        second_layer_ds[n] = 0;
                    }
                    for(int n = 0; n < NUM_NEURONS; n++){
                        fully_connected_layer_ds[k][n] += ((first_layer[n] * soft_max_layer_ds[k]) / BATCH_SIZE);
                        second_layer_ds[n] += fully_connected_layer_w[k][n] * soft_max_layer_ds[k];
                    }
                }

                for(int n = 0; n < NUM_NEURONS; n++){
                    for(int r = 0; r < ROWS; r++){
                        for(int c = 0; c < COLS; c++){
                            input_layer_ds[n][r][c] +=  (current_image[r][c] * second_layer_ds[n])/BATCH_SIZE;
                        }
                    }
                }

                //UPDATE WEIGHTS
                //copy to contiguous array to copy to CUDA mem
                for(int k = 0; k < (int) NUM_NEURONS/EPOCH_SIZE; k++){
                    for(int n = 0; n < NUM_NEURONS; n++){
                        hidden_layer_w2[k*NUM_NEURONS + n] = fully_connected_layer_w[k][n] ;
                        hidden_layer_ds2[k*NUM_NEURONS + n] = fully_connected_layer_ds[k][n];
                    }
                }
                hipMemcpy(dense_layer_w2, hidden_layer_w2, NUM_NEURONS* ((int)NUM_NEURONS/EPOCH_SIZE) *(sizeof(float)), hipMemcpyHostToDevice);
                hipMemcpy(dense_layer_ds2, hidden_layer_ds2, NUM_NEURONS* ((int)NUM_NEURONS/EPOCH_SIZE) *(sizeof(float)), hipMemcpyHostToDevice);

                update_dense_weights<<<(int)NUM_NEURONS/EPOCH_SIZE, NUM_NEURONS>>>(dense_layer_w2, dense_layer_ds2);

                //copy back
                hipMemcpy(hidden_layer_w2, dense_layer_w2, NUM_NEURONS* ((int)NUM_NEURONS/EPOCH_SIZE) *(sizeof(float)), hipMemcpyHostToDevice);
                hipMemcpy(hidden_layer_ds2, dense_layer_ds2, NUM_NEURONS* ((int)NUM_NEURONS/EPOCH_SIZE) *(sizeof(float)), hipMemcpyHostToDevice);


                for(int k = 0; k < (int) NUM_NEURONS/EPOCH_SIZE; k++){
                    for(int n = 0; n < NUM_NEURONS; n++){
                        fully_connected_layer_w[k][n] = hidden_layer_w2[k*NUM_NEURONS + n];
                        fully_connected_layer_ds[k][n] = hidden_layer_ds2[k*NUM_NEURONS + n];
                    }
                }

                /*----------------------*/

                for(int n = 0; n < NUM_NEURONS; n++){
                    for(int r = 0; r < ROWS; r++){
                        for(int c = 0; c < COLS; c++){
                            hidden_layer_w1[n*ROWS*COLS + r*ROWS + c] =  input_layer_w[n][r][c];
                            //std::cout << input_layer_ds[n][r][c] << std::endl;
                            //std::cout << input_layer_w[n][r][c] << std::endl;
                            hidden_layer_ds1[n*ROWS*COLS + r*ROWS + c] = input_layer_ds[n][r][c];
                        }
                    }
                }
                hipMemcpy(dense_layer_w1, hidden_layer_w1,  NUM_NEURONS*ROWS*COLS*(sizeof(float)), hipMemcpyHostToDevice);
                hipMemcpy(dense_layer_ds1, hidden_layer_ds1, NUM_NEURONS*ROWS*COLS*(sizeof(float)), hipMemcpyHostToDevice);

                update_dense_weights<<<(int)NUM_NEURONS/EPOCH_SIZE, NUM_NEURONS>>>(dense_layer_w1, dense_layer_ds1);

                //copy back
                hipMemcpy(hidden_layer_w1, dense_layer_w1, NUM_NEURONS* ((int)NUM_NEURONS/EPOCH_SIZE) *(sizeof(float)), hipMemcpyHostToDevice);
                hipMemcpy(hidden_layer_ds1, dense_layer_ds1, NUM_NEURONS* ((int)NUM_NEURONS/EPOCH_SIZE) *(sizeof(float)), hipMemcpyHostToDevice);


                for(int n = 0; n < NUM_NEURONS; n++){
                    for(int r = 0; r < ROWS; r++){
                        for(int c = 0; c < COLS; c++){
                            input_layer_w[n][r][c] = hidden_layer_w1[n*ROWS*COLS + r*ROWS + c];
                            //std::cout << input_layer_ds[n][r][c] << std::endl;
                            //std::cout << input_layer_w[n][r][c] << std::endl;
                            input_layer_ds[n][r][c] = hidden_layer_ds1[n*ROWS*COLS + r*ROWS + c];
                        }
                    }
                }
            }
            if(j % 100 == 0){
                //std::cout << input_layer_w[0][0][0] << std::endl;
                printf("Epoch %d: Round %d: accuracy=%f\n", e, j, correct/total);
            }

        }
    }

    return 0;

}























//
